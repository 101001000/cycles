
#include <hip/hip_runtime.h>
/* SPDX-FileCopyrightText: 2011-2022 Blender Foundation
 *
 * SPDX-License-Identifier: Apache-2.0 */

/* CUDA kernel entry points */

#ifdef __CUDA_ARCH__

#  include "kernel/device/portablert/compat.h"
#  include "kernel/device/portablert/config.h"
#  include "kernel/device/portablert/globals.h"

#  include "kernel/device/gpu/image.h"
#  include "kernel/device/gpu/kernel.h"

#endif
